#include "hip/hip_runtime.h"

#include "AveragepoolLayer.h"

#include "../mpc/RSS.h"
#include "../mpc/TPC.h"
#include "../mpc/FPC.h"
#include "../mpc/OPC.h"

#include <numeric>

extern nlohmann::json piranha_config;

template<typename T, template<typename, typename...> typename Share>
Profiler AveragepoolLayer<T, Share>::averagepool_profiler;

template<typename T, template<typename, typename...> typename Share>
AveragepoolLayer<T, Share>::AveragepoolLayer(AveragepoolConfig* conf, int _layerNum, int seed) : Layer<T, Share>(_layerNum),
 	conf(conf->imageHeight, conf->imageWidth, conf->features, 
	  		conf->poolSize, conf->stride, conf->batchSize, conf->microBatchSize),
 	_activations(conf->batchSize * conf->features * 
			(((conf->imageWidth - conf->poolSize)/conf->stride) + 1) * 
 		    (((conf->imageHeight - conf->poolSize)/conf->stride) + 1)),
 	_deltas(conf->batchSize * conf->features * conf->imageHeight * conf->imageWidth) {
	// nothing
};

template<typename T, template<typename, typename...> typename Share>
void AveragepoolLayer<T, Share>::loadSnapshot(std::string path) {
    // do nothing
}

template<typename T, template<typename, typename...> typename Share>
void AveragepoolLayer<T, Share>::saveSnapshot(std::string path) {
    // do nothing
}

template<typename T, template<typename, typename...> typename Share>
void AveragepoolLayer<T, Share>::printLayer()
{
    std::cout << "----------------------------------------------" << std::endl;
    std::cout << "(" << this->layerNum+1 << ") Averagepool Layer\t  " << conf.imageHeight << " x " << conf.imageWidth 
		 << " x " << conf.features << std::endl << "\t\t\t  " 
		 << conf.poolSize << "  \t\t(Pooling Size)" << std::endl << "\t\t\t  " 
		 << conf.stride << " \t\t(Stride)" << std::endl << "\t\t\t  " 
		 << conf.batchSize << "\t\t(Batch Size)" << std::endl;
}

template<typename T, template<typename, typename...> typename Share>
void AveragepoolLayer<T, Share>::forward(const Share<T> &input, int micro_batch_idx) {

    if (piranha_config["debug_all_forward"]) {
        printf("layer %d\n", this->layerNum);
        //printShareTensor(*const_cast<Share<T> *>(&input), "fw pass input (n=1)", 1, 1, 1, input.size() / conf.batchSize);
    }

	LOG_S(1) << "Executing Averagepool.forward";

    this->layer_profiler.start();
    averagepool_profiler.start();
   
    // Get a "view" of the activation that is only relevant to the current micro batch.
    Share<T>* activations; 
    if(micro_batch_idx == -1) {
        activations = &_activations;
    } else {
        CHECK_F(input.size()  == MICRO_BATCH_SIZE * conf.imageHeight * conf.imageWidth * conf.features, "the size of input must be consistent with the number of elements per microbatch");
        int activation_microbatch_size = _activations.size() / conf.batchSize * MICRO_BATCH_SIZE;
        activations = new Share<T>(_activations, micro_batch_idx * activation_microbatch_size, (micro_batch_idx+1) * activation_microbatch_size); 
    }
    activations->zero();

    Share<T> pools((size_t)0);
    for(int share = 0; share < Share<T>::numShares(); share++) {
        gpu::averagepool_im2row(
                input.getShare(share),
                pools.getShare(share),
                conf.imageWidth, conf.imageHeight, conf.poolSize, conf.features, MICRO_BATCH_SIZE,
                conf.stride, 0
        );
    }

    for(int share = 0; share < Share<T>::numShares(); share++) {
        gpu::reduceSum(
            pools.getShare(share),
            activations->getShare(share),
            false, activations->size(), conf.poolSize * conf.poolSize
        );
    }

    dividePublic(*activations, (T)(conf.poolSize * conf.poolSize));

    this->layer_profiler.accumulate("averagepool-forward");
    averagepool_profiler.accumulate("averagepool-forward");

    if (piranha_config["debug_all_forward"]) {
        //printShareTensor(*const_cast<Share<T> *>(&activations), "fw pass activations (n=1)", 1, 1, 1, activations.size() / conf.batchSize);
        std::vector<double> vals(activations->size());
        copyToHost(*activations, vals);
        
        printf("avgpool,fw activation,min,%e,avg,%e,max,%e\n", 
                *std::min_element(vals.begin(), vals.end()),
                std::accumulate(vals.begin(), vals.end(), 0.0) / static_cast<float>(vals.size()), 
                *std::max_element(vals.begin(), vals.end()));
    }
}

template<typename T, template<typename, typename...> typename Share>
void AveragepoolLayer<T, Share>::backward(const Share<T> &delta, const Share<T> &forwardInput, int micro_batch_idx) {

    if (piranha_config["debug_all_backward"]) {
        printf("layer %d\n", this->layerNum);
        //printShareFinite(*const_cast<Share<T> *>(&delta), "input delta for bw pass (first 10)", 10);
        std::vector<double> vals(delta.size());
        copyToHost(
            *const_cast<Share<T> *>(&delta),
            vals
        );
        
        printf("avgpool,bw input delta,min,%e,avg,%e,max,%e\n", 
                *std::min_element(vals.begin(), vals.end()),
                std::accumulate(vals.begin(), vals.end(), 0.0) / static_cast<float>(vals.size()), 
                *std::max_element(vals.begin(), vals.end()));
    }

	LOG_S(1) << "Executing Averagepool.backward";

    this->layer_profiler.start();
    averagepool_profiler.start();
    Share<T>* deltas; 
    if(micro_batch_idx == -1) {
        deltas = &_deltas;
    } else {
        CHECK_F(delta.size() == _activations.size() / conf.batchSize * MICRO_BATCH_SIZE, "size of input delta is %i instead of %i", delta.size(), _activations.size() / conf.batchSize * MICRO_BATCH_SIZE);
        CHECK_F(forwardInput.size()  == MICRO_BATCH_SIZE * conf.imageHeight * conf.imageWidth * conf.features, "the size of forwardInput must be consistent with the mumber of elements per microbatch");
        int size_per_micro_batch = conf.imageHeight * conf.imageWidth * conf.features * MICRO_BATCH_SIZE;
        deltas = new Share<T>(_deltas, micro_batch_idx * size_per_micro_batch, (micro_batch_idx + 1) * size_per_micro_batch); 
    }
    deltas->zero();
    CUDA_CHECK(hipStreamSynchronize(0));
    for (int share = 0; share < Share<T>::numShares(); share++) {
        gpu::averagepool_expand_delta(delta.getShare(share), deltas->getShare(share),
                (int)conf.features, (int)(conf.poolSize * conf.poolSize));
    }
    CUDA_CHECK(hipStreamSynchronize(0));

    dividePublic(*deltas, (T)(conf.poolSize * conf.poolSize));
    CUDA_CHECK(hipStreamSynchronize(0));

    averagepool_profiler.accumulate("averagepool-backward");
    this->layer_profiler.accumulate("averagepool-backward");
}

template class AveragepoolLayer<uint32_t, RSS>;
template class AveragepoolLayer<uint64_t, RSS>;

template class AveragepoolLayer<uint32_t, TPC>;
template class AveragepoolLayer<uint64_t, TPC>;

template class AveragepoolLayer<uint32_t, FPC>;
template class AveragepoolLayer<uint64_t, FPC>;

template class AveragepoolLayer<uint32_t, OPC>;
template class AveragepoolLayer<uint64_t, OPC>;

